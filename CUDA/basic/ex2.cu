
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void KernelFunction(int a, int b, int c)
{
    int sum;

    sum = (a + b + c);

    return;
}

int main()
{
    /*  <<<BLOCK, THREAD>>>, total threads: (BLOCK * THREAD)    */
    KernelFunction<<<6, 6>>>(1, 2, 3);
    puts("Successfully called threads");

    return 0;
}
