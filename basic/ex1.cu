/*
 *  A example of use the memory of a graphics card.
 */

#include <hip/hip_runtime.h>
#include <stdio.h>

int main()
{
    int i;
    int input[5]    = {1, 2, 3, 4, 5};
    int output[5]   = {0};
    int *graphicsCardMem;

    /*  cudaMalloc a device array   */
    hipMalloc((void **) &graphicsCardMem, (5 * sizeof(int)));  
    
    /*  copy datas from the PC to the graphics card  */
    hipMemcpy(graphicsCardMem, input, (5 * sizeof(int)), hipMemcpyHostToDevice);

    /*  copy datas from the graphics card to the PC */
    hipMemcpy(output, graphicsCardMem, (5 * sizeof(int)), hipMemcpyDeviceToHost);

    for (i = 0; i < 5; ++i) {
        printf("[%d] output value: %d\n", i, output[i]);
    }
    /*  free memory */
    hipFree(graphicsCardMem);
    
    return 0;
}
